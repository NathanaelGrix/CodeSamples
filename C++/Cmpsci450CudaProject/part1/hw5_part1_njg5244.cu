//Nathanael Grix
//Hw5 Cuda Core Benchmark:
//Purpose: Compare cpu and gpu execution time to sum large arrays of numbers on
//	Penn State's computing cluster. I will compare the exectution time from 
//	sets of 1 to 200,000,000.

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define B 1024

//unused implementation
__global__ void scan(float *g_odata, float *g_idata, unsigned long long int n);
//first part of sacning a large array
__global__ void prescan(float *g_odata, float *g_idata, unsigned long long int n, float* blkSum, unsigned long int N);
//last part of scaning a large array
__global__ void postscan(float *g_odata, unsigned long long int n, float* blkSum);

//cpu impementation of a simple scan
void scanCPU(float *f_out, float *f_in, unsigned long long int i_n);

//function for getting the time difference
double myDiffTime(struct timeval &start, struct timeval &end)
{
	double d_start, d_end;
	d_start = (double)(start.tv_sec + start.tv_usec/1000000.0);
	d_end = (double)(end.tv_sec + end.tv_usec/1000000.0);
	return (d_end - d_start);
}

int main(int argc,char** argv) {

	//number of values and threads
	unsigned long int N;

	//default val or not?
	if(argc==2){
		N=atol(argv[1]);
	}
	else{
		N=100000000;
	}

	printf("N=%d, B=%d\n\n",N,B);
	//calculate the number of blocks
	unsigned long long int blkNum =(unsigned long long int)ceil((double)N/((double)(B*2)));
	
	//the values used to calculate the scans
	float *a, *c, *g, *blkSumin, *blkOut;
	//values used to time 
	timeval start, end;
	
	//variables used to talk to memory from cuda
	float *dev_a, *dev_g, *dev_blkSum;
	//sizes of an array of N and the sum of the blocks
	unsigned long long int size = N * sizeof(float);
	unsigned long long int blkSumSize = blkNum * sizeof(float);
	
	//time vars
	double d_gpuTime, d_cpuTime, d_gpuLoadT, d_gpuPreT, d_gpuScaT, d_gpuPostT;
	double load[4];

	//the serial final result
	c=(float*)malloc(size);

	//cuda optomized malloc for vars that cuda will need
	hipHostMalloc((void **) &a, size, hipHostMallocDefault);
	hipHostMalloc((void **) &g, size, hipHostMallocDefault);
	hipHostMalloc((void **) &blkSumin, blkSumSize, hipHostMallocDefault);
	hipHostMalloc((void **) &blkOut, blkSumSize, hipHostMallocDefault);
	
	// initialize matrices a 
	for (unsigned long long int i = 0; i < N; i++)
	{	
		// fill a with rand floats
		a[i] = (float)(rand() % 1000000) / 1000.0;
	}
	// initialize a and b matrices here
	hipMalloc((void **) &dev_a, size);
	hipMalloc((void **) &dev_g, size);
	hipMalloc((void **) &dev_blkSum, blkSumSize);
	
	//		START TIME
	gettimeofday(&start, NULL);
	
	//		COPY 1 TIME
	// fill the graphics card's memory with a
	hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
	gettimeofday(&end, NULL);
	load[0] = myDiffTime(start, end);

	//		PRE TIME
	// have small scans done on the memory in blocks
	gettimeofday(&start, NULL);
	prescan<<<blkNum,B,2*B*sizeof(float)>>>(dev_g, dev_a, B*2, dev_blkSum, N);
	hipDeviceSynchronize();
	gettimeofday(&end, NULL);
	d_gpuPreT = myDiffTime(start, end);

	//		COPY 2 TIME
	// get the blocks' sums out of the graphics cards so we can do a small serial
	// 	scan on them.
	gettimeofday(&start, NULL);
	hipMemcpy(blkSumin, dev_blkSum, blkSumSize, hipMemcpyDeviceToHost);
	gettimeofday(&end, NULL);
	load[1] += myDiffTime(start, end);
	
	//		SCAN TIME
	// do a small serial scan on the blocks' sums.
	gettimeofday(&start, NULL);
	scanCPU(blkOut, blkSumin, blkNum);
	gettimeofday(&end, NULL);
	d_gpuScaT = myDiffTime(start, end);

	//		COPY 3 TIME
	// give the blocks back to the graphics cards for post.
	gettimeofday(&start, NULL);
	hipMemcpy(dev_blkSum, blkOut, blkSumSize, hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	gettimeofday(&end, NULL);
	load[2] += myDiffTime(start, end);

	//		POST TIME
	// add the new scanned sum to each block to get the scan of the whole array
	gettimeofday(&start, NULL);
	postscan<<<(unsigned long long int)ceil((double)N/(double)(B)),B>>>(dev_g, N, dev_blkSum);
	gettimeofday(&end, NULL);
	d_gpuPostT = myDiffTime(start, end);

	//		COPY 4 TIME
	// finally get the result into normal memory
	gettimeofday(&start, NULL);
	hipMemcpy(g, dev_g, size, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	
	//		END TIME
	gettimeofday(&end, NULL);
	load[3] += myDiffTime(start, end);

	// get the total time for the loads
	d_gpuLoadT = load[0] + load[1] + load[2] +load[3];
	// get the total time for my implementation
	d_gpuTime = d_gpuLoadT + d_gpuPreT + d_gpuScaT + d_gpuPostT;

	//do a serial scan on the whole array
	gettimeofday(&start, NULL);
	scanCPU(c, a, N);
	
	gettimeofday(&end, NULL);
	d_cpuTime = myDiffTime(start, end);
	
	// free the memory for the graphics card
	hipFree(dev_a); hipFree(dev_g); hipFree(dev_blkSum);

	// display at least 50 sums and compare the error percent of the serial
	// 	vs cuda implentation.
	for (unsigned long long int i = 0; i < N; i++)
	{	if(N>50){
			if(i%(N/50)==0 || i == N-1){
				printf("c[%i] = %0.3f, g[%i] = %0.3f\n", i, c[i], i, g[i]);
				printf("error percent = %f%\n",fabs((g[i]-c[i])/c[i])*100.0);
			}
		}
		else{
			printf("c[%i] = %0.3f, g[%i] = %0.3f\n", i, c[i], i, g[i]);
			printf("error percent = %f%\n",fabs((g[i]-c[i])/c[i])*100.0);
		}
	}
	
	// print out the time for the cpu and the gpu
	printf("CPU Time for scan size %i: %f\n", N, d_cpuTime);
	printf("GPU Time for scan size %i: %f\n", N, d_gpuTime);
	// print out the times of the components of the gpu implementation
	printf("\tgpu Load=%f, Pre=%f, Sca=%f, Post=%f\n",d_gpuLoadT, d_gpuPreT, d_gpuScaT, d_gpuPostT);
	// print out the various loading times for the implementation
	printf("\t\tgpu load times={%f,%f,%f,%f}\n", load[0],load[1],load[2],load[3]);

	// free c from memory
	free(c);
	
	
	// free the rest of the variables
	hipHostFree(a);
	hipHostFree(g);
	hipHostFree(blkSumin);
	hipHostFree(blkOut);
}

// old unused verson of scan that was included in the naivescan
__global__ void scan(float *g_odata, float *g_idata, unsigned long long int n)
{
	extern __shared__ float temp[]; // allocated on invocation
	unsigned long long int thid = threadIdx.x;
	unsigned long long int pout = 0, pin = 1;
	// Load input into shared memory.
	// This is exclusive scan, so shift right by one
	// and set first element to 0
	temp[pout*n + thid] = (thid > 0) ? g_idata[thid-1] : 0;
	__syncthreads();
	for (unsigned long long int offset = 1; offset < n; offset *= 2)
	{
		pout = 1 - pout; // swap double buffer indices
		pin = 1 - pout;
		if (thid >= offset)
			temp[pout*n+thid] += temp[pin*n+thid - offset];
		else
			temp[pout*n+thid] = temp[pin*n+thid];

		__syncthreads();
	}
	g_odata[thid] = temp[pout*n+thid]; // write output
}

// function that adds the new blocks' sums back into the scanned blocks
__global__ void postscan(float *g_odata, unsigned long long int n, float* blkSum){
	unsigned long long int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n) 
	{	// prescan does two vals per block but this one does 1 so 
		// 	adjust the block ID to reflect that.
		g_odata[i] += blkSum[blockIdx.x/2];
	}
}

// scans a block of the data
__global__ void prescan(float *g_odata, float *g_idata, unsigned long long int n, float* blkSum, unsigned long int N) 
{ 
	// name the newly allocated memory as temp.
	extern  __shared__  float temp[];
	
	//get the begining of the array for this block
	unsigned long long int bldata = blockIdx.x*(blockDim.x);

	// get the thread's id for the block
	unsigned long long int thid = threadIdx.x; 
	
	// initialize the offset to 1
	unsigned long long int offset = 1;

	// fill the temp array with the input data because the math can be executed
	// 	much faster than other forms of meory
	temp[2*thid]   = 0.0; 
    	temp[2*thid+1] = 0.0;
	if(2*(thid+bldata)<N){
		temp[2*thid]   = g_idata[2*(thid+bldata)]; 
	}
	if(2*(thid+bldata)+1<N){
    		temp[2*thid+1] = g_idata[2*(thid+bldata)+1];
	}
	
	//log for loop
	for (unsigned long long int d = n>>1; d > 0; d >>= 1) 
	// build sum in place up the tree 
    	{ 
        	__syncthreads(); 
		if (thid < d){    
        		unsigned long long int ai = offset*(2*thid+1)-1; 
			unsigned long long int bi = ai+offset; 
		    	temp[bi] += temp[ai];
  		} 
		// exponential offset
        	offset <<= 1; 
    	}
 
	// set the lst value and get the total sum
	if (thid == 0){
		blkSum[blockIdx.x] = temp[n-1];
		temp[n-1] = 0;
	}
 
	for (unsigned long long int d = 1; d < n; d <<= 1) 
	// traverse down tree & build scan 
    	{ 
        	offset >>= 1; 
        	__syncthreads(); 
		if (thid < d){
			unsigned long long int ai = offset*(2*thid+1)-1; 
			unsigned long long int bi = ai+offset; 
			float t   = temp[ai]; 
            		temp[ai]  = temp[bi]; 
            		temp[bi] += t; 
        	} 
    	}
	__syncthreads();

	// write results to device memory 
	if(2*(thid+bldata)<N){
		g_odata[2*(thid+bldata)]   = temp[2*thid];
	}
	if(2*(thid+bldata)+1<N){
		g_odata[2*(thid+bldata)+1] = temp[2*thid+1];
	}
}
 
void scanCPU(float *f_out, float *f_in, unsigned long long int i_n)
{
	f_out[0] = 0;
	for (unsigned long long int i = 1; i < i_n; i++)
		f_out[i] = f_out[i-1] + f_in[i-1];

}
